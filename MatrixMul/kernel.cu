#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

const int SIZE = 150;

hipError_t cudaStatus;

__global__ void matrixMul(int* c, const int* a, const int* b)
{	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < SIZE && j < SIZE)
	{
		int temp = 0;
		for (int k = 0; k < SIZE; k++)
		{
			temp += a[i * SIZE + k] * b[k * SIZE + j];
		}
		c[i * SIZE + j] = temp;
	}
}

int main()
{	// Square Matrix Multiplication

	int a[SIZE][SIZE];
	int b[SIZE][SIZE];
	int c[SIZE][SIZE] = { {0} };

	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			a[i][j] = 1;
			b[i][j] = 1;
		}
	}
	

	// Device memory allocation
	int* dev_a;
	int* dev_b;
	int* dev_c;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	cudaStatus = hipMalloc((void**)&dev_a, SIZE * SIZE * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_b, SIZE * SIZE * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_c, SIZE * SIZE * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	hipMemset(dev_c, 0, SIZE * SIZE * sizeof(int));

	cudaStatus = hipMemcpy(dev_a, a, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	cudaStatus = hipMemcpy(dev_b, b, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks((SIZE + 31) / 32, (SIZE + 31) / 32); 

	// Kernel call
	matrixMul << <numBlocks, threadsPerBlock >> > (dev_c, dev_a, dev_b);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA error after kernel launch: %s\n", hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	cudaStatus = hipMemcpy(c, dev_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	
	// Free device memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);



	// Displaying the result
	printf("Matrix C:\n");
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			printf("%d ", c[i][j]);
		}
		printf("\n");
	}

}